#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// Função cuda que realiza o produto de vetores na gpu:
__global__ void multiply(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] * b[index];
        if(n<=10)
            printf("Thread %d executando, multiplicando %d * %d = %d\n", index, a[index], b[index], c[index]);
    }
}

// Função equivalente na cpu:
void mult_cpu(int *a, int *b, int *c, int n) {
    for (int i = 0; i < n; ++i) {
        c[i] = a[i] * b[i];
    }
}

int main() {
    // Variável que guarda as dimensões do vetor
    int n;

    std::cout << "Digite o tamanho desejado para os vetores: " << std::endl;
    std::cin >> n;

    // Declaração dos vetores cpu e gpu
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    // Alocação de memória no dispositivo (GPU)
    hipMalloc(&d_a, n * sizeof(int));
    hipMalloc(&d_b, n * sizeof(int));
    hipMalloc(&d_c, n * sizeof(int));

    // Alocação de memória na CPU
    a = new int[n];
    b = new int[n];
    c = new int[n];

    // Inicialização dos vetores a e b
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i;
    }

    if(n<=10){
        // Imprimir os vetores originais
        std::cout << "Vetor a: [";
        for (int i = 0; i < n; i++) {
            if(i<n-1)
                std::cout << a[i] << ", ";
            else
                std::cout << a[i];
        }
        std::cout << "]" << std::endl;

        std::cout << "Vetor b: [";
        for (int i = 0; i < n; i++) {
            if(i<n-1)
                std::cout << b[i] << ", ";
            else
                std::cout << b[i];
        }
        std::cout << "]" << std::endl;
    }

    // Copiar dados da CPU para a GPU
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Configurar a execução do kernel
    int blockSize = 1;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Captura o tempo atual antes da execução
    auto start = std::chrono::high_resolution_clock::now();

    // Chamar o kernel
    std::cout << "Chamando o kernel com " << numBlocks << " blocos e " << blockSize << " threads por bloco\n";
    multiply<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    // Copiar o resultado de volta para a CPU
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Captura o tempo após a execução
    auto end = std::chrono::high_resolution_clock::now();

    // Calcula a duração, ou seja, o tempo decorrido
    std::chrono::duration<double> elapsed = end - start;

    // Converte a duração para segundos e imprime
    std::cout << "Tempo decorrido pelo processamento feito em cuda: " << elapsed.count() << " segundos" << std::endl;

    if(n<=10){
        // Imprimir o vetor resultado
        std::cout << "Vetor resultado: [";
        for (int i = 0; i < n; i++) {
            if(i<n-1)
                std::cout << c[i] << ", ";
            else
                std::cout << c[i];
        }
        std::cout << "]" << std::endl;
    }

    // Captura o tempo atual antes da execução
    auto start_cpu = std::chrono::high_resolution_clock::now();

    // Chamar a função de soma em CPU
    mult_cpu(a, b, c, n);

    // Captura o tempo após a execução
    auto end_cpu = std::chrono::high_resolution_clock::now();

    // Calcula a duração, ou seja, o tempo decorrido
    std::chrono::duration<double> elapsed_cpu = end_cpu - start_cpu;

    // Converte a duração para segundos e imprime
    std::cout << "Tempo decorrido pelo processamento feito em CPU: " << elapsed_cpu.count() << " segundos" << std::endl;

    // Liberar memória alocada
    delete[] a;
    delete[] b;
    delete[] c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
